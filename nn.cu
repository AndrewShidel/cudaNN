
#include <hip/hip_runtime.h>
#include <vector>
#include <string>
#include <iostream>
#include <algorithm>
#include <sstream>
#include <ctime>
#include <signal.h>
#include <unistd.h>
#include <stdio.h>
#include <fstream>
#include <stdlib.h>
#include <math.h>
#include <unordered_set>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;
struct Edge;
struct Vertex;
struct Layer;
int random(int min, int max);
float average(float average, float dataPoint);
void setupListeners();

template<typename T>
inline void removeFromVector(vector<T> & v, const T & item);

struct Edge {
    float weight;
    float change;
    Vertex* from;
    Vertex* to;
};

struct Vertex {
    vector<Edge*> inputs;
    vector<Edge*> outputs;
    float output;
    float bias;
    int layer;
    bool isOutput;
    int index;
    float delta;
    float error;
};

struct Layer {
    vector<Vertex*> nodes;
    vector<Edge*> edges;
    int* edgeNodeMapping;
};

class NN {
public:
    NN(bool useGPU, int input, vector<int> hidden, int output);

    vector<Layer> layers;
    vector<Vertex*> nodes;
    vector<Edge*> edges;
    int inputSize;
    int outputSize;
    int outputIdx;
    bool useGPU;
    int nodeIdx;

    Vertex* addVertex(std::vector<Vertex*> inputs, std::vector<Vertex*> outputs);
    Vertex* addVertex(int inputCount, int outputCount, int layer, bool useBasicLayering);
    Vertex* addVertex(int inputs, int outputs, bool isInput, bool isOutput, int layer, bool useStrictLayers);
    void removeVertex();
    Edge* addEdge(Vertex* from, Vertex* to, bool addLayer);
    Edge* addEdge(Vertex* from, Vertex* to);
    void removeEdge(Edge* edge);
    int findLayer(Vertex* vertex);
    void initRNGCPU();
    float trainGPU(vector<float> inputs, vector<float> target);
    float trainGPU(vector<float> inputs, vector<float> target, float learningRate, float momentum);
    vector<float> runGPULauncher(vector<float>& inputs);
    vector<float> runCpu(vector<float>& inputs);
    vector<float> run(vector<float>& inputs);
    void updateHostMemory();
    void updateDeviceMemory();
    void print(ostream& output);
    double layerDist(double x, int mean);

    // CUDA pointers
    float* d_weights;
    float* d_outputs;
    float* d_bias;
    int* d_edgeNodeMappingTo;
    int* d_edgeNodeMappingFrom;
    float* d_errors;
    float* d_deltas;
    float* d_changes;
    float* d_target;
    bool* d_managementThreads;
    hiprandState_t* RNGStates;
};

bool userEntryMode = false;
bool running = true;
void my_handler(int s){
    printf("Caught signal %d. Entering user input mode.\n",s);
    if (userEntryMode) {
        running = false;
    }
    userEntryMode = true;
    setupListeners();
}
void setupListeners() {
    struct sigaction sigIntHandler;

    sigIntHandler.sa_handler = my_handler;
    sigemptyset(&sigIntHandler.sa_mask);
    sigIntHandler.sa_flags = 0;

    sigaction(SIGINT, &sigIntHandler, NULL);
}

/*int test(bool useGPU, vector<vector<float>> inputs, vector<vector<float>> targets, vector<int> hidden) {
    int inputSize = inputs[0].size();
    int outputSize = targets[0].size();
    srand(0);
    setupListeners();
    NN nn(useGPU, inputSize, hidden, outputSize);

    //ofstream outputFile(string(useGPU?"gpu":"cpu") + ".graph", ofstream::out);
    //nn.print(outputFile);

    clock_t begin = clock();
    vector<float> target(outputSize);
    vector<float> input(inputSize);

    float error = 1.0;

    do {
        int index = random(0, inputSize);
        if (userEntryMode) {
            std::cout << "\n";
            cin >> iInput;
            std::cout << "\n";
        }
    } while(error > 0 && running);
}*/

int test(bool useGPU, int inputSize, vector<int> hidden, int outputSize) {
    srand(0);
    setupListeners();
    NN nn(useGPU, inputSize, hidden, outputSize);
    ofstream outputFile(string(useGPU?"gpu":"cpu") + ".graph", ofstream::out);
    nn.print(outputFile);

    clock_t begin = clock();

    vector<float> target(outputSize);
    vector<float> input(inputSize);
    float error = 1.0;
    do {
        int iInput = random(100, 1000);
        if (userEntryMode) {
            std::cout << "\n";
            cin >> iInput;
            std::cout << "\n" << iInput << "\n";
        }
        int n = iInput;
        int i = 0;
        while (n) {
            input[i++] = (n % 10)/10.0;
            n /= 10;
        }

        //input[0] = iInput/1000.0;
        target[0] = iInput%2==0?1.0:0.0;
        //target[0] = (iInput>500?1.0:0.0);

        std::cout << "Expected: " << target[0] << ", Input: " << iInput << ", ";
        if (userEntryMode) {
            float result = nn.runGPULauncher(input)[0];
            cout << "Output: " << result << "\n";
        }else{
            error = average(error, nn.trainGPU(input, target, error*2, error));
            std::cout << "\rError: " << error;
        }
    } while(error > 0.005 && running);

    clock_t end = clock();
    double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
    cout << "Time ms: " << elapsed_secs*1000 << "\n";
    return 0;
}

int main(int argc, char** argv) {
    vector<int> hiddenSizes;
    hiddenSizes.push_back(5);
    hiddenSizes.push_back(10);
    hiddenSizes.push_back(50);
    hiddenSizes.push_back(200);
    hiddenSizes.push_back(50);
    hiddenSizes.push_back(10);
    //hiddenSizes.push_back(1);

    for (int i=0; i<1; i++) {
      test(true, 3, hiddenSizes, 1);
    }
    cout << "--------------------------\n";
    /*for (int i=0; i<1; i++) {
      test(false, 1, hiddenSizes, 1);
    }*/
}

// layers: [inputSize, hiddenSize, outputSize]
NN::NN(bool useGPU, int input, vector<int> hidden, int output) {
    // comment in for release
    //srand(time(NULL));
    inputSize = 0;
    outputSize = 0;
    outputIdx = 0;
    nodeIdx = 0;
    this->useGPU = useGPU;

    int nodes = 0;
    for (int i=0; i<input; ++i) {
        addVertex(vector<Vertex*>(), vector<Vertex*>());
        //addVertex(0, 0, true, false, 0, false);
        nodes++;
        inputSize++;
    }
    for (int i=0; i<hidden.size(); ++i) {
        int prevLayerNodes = nodes;
        for (int j=0; j<hidden[i]; ++j) {
            addVertex(random(1, prevLayerNodes), 0, i+1, false);
            //addVertex(random(1, prevLayerNodes), 0, false, false, i+1, false);
            nodes++;
        }
    }

    for (int i=0; i<output; ++i) {
        addVertex(random(1, nodes), 0, hidden.size()+1, false);
        //addVertex(random(1, nodes), 0, false, true, 2, false);
        outputSize++;
    }

    /*for (int i=0; i<layers.size()-1; ++i) {
        for (int j=0; j<layers[i].nodes.size(); ++j) {
            Vertex* vertex = layers[i].nodes[j];
            if (vertex->outputs.size() == 0) {
                Layer& outputLayer = layers[i+1];
                Vertex* outputVertex = outputLayer.nodes[
                    random(0,
                            outputLayer.nodes.size()-1
                    )
                ];
                addEdge(vertex, outputVertex, true);
            }
        }
    }*/


    if (useGPU) {
        updateDeviceMemory();
    }
}

double NN::layerDist(double x, int mean) {
    return -1*pow(2*x-mean,2)+mean;
}

void NN::updateDeviceMemory() {
    initRNGCPU();
    int edgeCount = edges.size();
    int nodeCount = nodes.size();

    size_t floatEdge = sizeof(float)*edgeCount;
    size_t floatNode = sizeof(float)*nodeCount;
    size_t intEdge = sizeof(int)*edgeCount;

    // TODO Free previous device memory

    // Malloc device memory
    hipMalloc(&d_weights, floatEdge);
    hipMalloc(&d_outputs, floatNode);
    hipMalloc(&d_bias, floatNode);
    hipMalloc(&d_edgeNodeMappingTo, intEdge);
    hipMalloc(&d_edgeNodeMappingFrom, intEdge);
    hipMalloc(&d_errors, floatNode);
    hipMalloc(&d_deltas, floatNode);
    hipMalloc(&d_changes, floatEdge);
    hipMalloc(&d_target, sizeof(float)*outputSize);
    hipMalloc(&d_managementThreads, sizeof(bool)*edgeCount);

    hipMemset(d_outputs, 0, floatNode);
    hipMemset(d_changes, 0, floatEdge);

    float* weights = (float*) malloc(floatEdge);
    float* bias = (float*) malloc(floatNode);
    int* edgeNodeMappingTo = (int*) malloc(intEdge);
    int* edgeNodeMappingFrom = (int*) malloc(intEdge);
    float* errors = (float*) malloc(floatNode);
    float* deltas = (float*) malloc(floatNode);
    float* changes = (float*) malloc(floatEdge);
    bool* managementThreads = (bool*) malloc(sizeof(bool)*edgeCount);

    int currEdge = 0;
    int currNode = 0;
    for (int i=0; i<layers.size(); ++i) {
        int edgesInLayer = 0;
        for (int j=0; j<layers[i].nodes.size(); ++j) {
            Vertex* node = layers[i].nodes[j];

            bias[currNode] = node->bias;
            errors[currNode] = node->error;
            deltas[currNode] = node->delta;

            for (int k=0; k<node->inputs.size(); ++k) {
                edgesInLayer++;
                Edge* edge = node->inputs[k];
                weights[currEdge] = edge->weight;
                changes[currEdge] = edge->change;
                managementThreads[currEdge] = (k==0);

                edgeNodeMappingTo[currEdge] = currNode;
                edgeNodeMappingFrom[currEdge] = edge->from->index;
                currEdge++;
            }
            int inputCount = node->inputs.size();
            node->index = currNode;
            currNode++;
        }
        cout << "Edges In Layer " << i << " = " << edgesInLayer << "\n";
    }

    for (int i=0; i<edgeCount; ++i) {
        if (edgeNodeMappingTo[i] < 0 || edgeNodeMappingTo[i]>nodeCount)
            cout << "edgeNodeMappingTo: [" << i << "]: " << edgeNodeMappingTo[i] << "\n";
        if (edgeNodeMappingFrom[i] < 0 || edgeNodeMappingFrom[i]>nodeCount)
            cout << "edgeNodeMappingFrom: [" << i << "]: " << edgeNodeMappingFrom[i] << "\n";
        if (weights[i] < 0 || weights[i]>1)
            cout << "weights: [" << i << "]: " << weights[i] << "\n";
    }
    for (int i=0; i<edgeCount; ++i) {
        if (edgeNodeMappingTo[i] < 0 || edgeNodeMappingTo[i]>nodeCount)
            cout << "edgeNodeMappingTo: [" << i << "]: " << edgeNodeMappingTo[i] << "\n";
    }

    hipMemcpy(d_weights, weights, floatEdge, hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, floatNode, hipMemcpyHostToDevice);
    hipMemcpy(d_edgeNodeMappingTo, edgeNodeMappingTo, intEdge, hipMemcpyHostToDevice);
    hipMemcpy(d_edgeNodeMappingFrom, edgeNodeMappingFrom, intEdge, hipMemcpyHostToDevice);
    hipMemcpy(d_errors, bias, floatNode, hipMemcpyHostToDevice);
    hipMemcpy(d_deltas, bias, floatNode, hipMemcpyHostToDevice);
    hipMemcpy(d_changes, bias, floatNode, hipMemcpyHostToDevice);
    hipMemcpy(d_managementThreads, managementThreads, sizeof(bool)*edgeCount, hipMemcpyHostToDevice);

    free(weights);
    free(bias);
    free(edgeNodeMappingTo);
    free(edgeNodeMappingFrom);
    free(errors);
    free(deltas);
    free(changes);
    free(managementThreads);
}

void NN::updateHostMemory() {
    int edgeCount = edges.size();
    int nodeCount = nodes.size();

    size_t floatEdge = sizeof(float)*edgeCount;
    size_t floatNode = sizeof(float)*nodeCount;
    size_t intEdge = sizeof(int)*edgeCount;
    //size_t intNode = sizeof(int)*nodeCount;

    float* weights = (float*) malloc(floatEdge);
    float* bias = (float*) malloc(floatNode);
    int* edgeNodeMappingTo = (int*) malloc(intEdge);
    int* edgeNodeMappingFrom = (int*) malloc(intEdge);
    float* errors = (float*) malloc(floatNode);
    float* deltas = (float*) malloc(floatNode);
    float* changes = (float*) malloc(floatEdge);

    hipMemcpy(weights, d_weights, floatEdge, hipMemcpyDeviceToHost);
    hipMemcpy(bias, d_bias, floatNode, hipMemcpyDeviceToHost);
    hipMemcpy(edgeNodeMappingTo, d_edgeNodeMappingTo, intEdge, hipMemcpyDeviceToHost);
    hipMemcpy(edgeNodeMappingFrom, d_edgeNodeMappingFrom, intEdge, hipMemcpyDeviceToHost);
    hipMemcpy(errors, d_errors, floatNode, hipMemcpyDeviceToHost);
    hipMemcpy(deltas, d_deltas, floatNode, hipMemcpyDeviceToHost);
    hipMemcpy(changes, d_changes, floatNode, hipMemcpyDeviceToHost);

    unordered_set<int> nodesCreated;
    vector<Vertex*> emptyEdgeList;
    for (int i=0; i<edgeCount; ++i) {
        int nodeTo = edgeNodeMappingTo[i];
        int nodeFrom = edgeNodeMappingFrom[i];

        int nodeToCreate = -1;
        if (!nodesCreated.count(nodeTo)) {
            nodeToCreate = nodeTo;
        }else if (!nodesCreated.count(nodeFrom)) {
            nodeToCreate = nodeFrom;
        }

        if (nodeToCreate != -1) {
            nodesCreated.insert(nodeToCreate);
            Vertex* vertex = addVertex(emptyEdgeList, emptyEdgeList);
            vertex->delta = deltas[nodeToCreate];
            vertex->error = errors[nodeToCreate];
            //vertex->change = changes[nodeToCreate];
            vertex->bias = bias[nodeToCreate];
        }
    }

    free(weights);
    free(bias);
    free(edgeNodeMappingTo);
    free(edgeNodeMappingFrom);
    free(errors);
    free(deltas);
    free(changes);
}

vector<float> NN::run(vector<float>& inputs) {
    if (useGPU) {
        return runGPULauncher(inputs);
    }else{
        return runCpu(inputs);
    }
}

__global__ void initRNG(unsigned int seed, hiprandState_t* states) {
    /* we have to initialize the state */
    hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
                blockIdx.x, /* the sequence number should be different for each core (unless you want all
                         cores to get the same sequence of numbers for some reason - use thread id! */
                0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
                &states[blockIdx.x]);
}

void NN::initRNGCPU() {
    int N = edges.size()/512+1;
    hipMalloc((void**) &RNGStates, N * sizeof(hiprandState_t));
    initRNG<<<N, 1>>>(/*time(0)*/0, RNGStates);
}

__global__ void runGPU(float* weights, int* edgeNodeMappingTo, int* edgeNodeMappingFrom, float* outputs, float* bias, bool* managementThreads, int offset, int n) {
    int id = blockIdx.x*blockDim.x+threadIdx.x + offset;
    if (id < n) {
        int nodeTo = edgeNodeMappingTo[id];
        int nodeFrom = edgeNodeMappingFrom[id];
        atomicAdd(&outputs[nodeTo], outputs[nodeFrom] * weights[id]);
        /*if (managementThreads[id]) {
            outputs[nodeTo] = 1/(1+exp(-1*(outputs[nodeTo]+bias[nodeTo])));
        }*/
    }
}

__global__ void runGPUPost(float* outputs, float* bias, int offset, int n) {
    int id = blockIdx.x*blockDim.x+threadIdx.x + offset;
    if (id < n) {
        outputs[id] = 1/(1+exp(-1*(outputs[id]+bias[id])));
    }
}

__global__ void runGPUProb(float* weights, int* edgeNodeMappingTo, int* edgeNodeMappingFrom, float* outputs, float* bias, int offset, int n, hiprandState_t* states) {
    int id = blockIdx.x*blockDim.x+threadIdx.x + offset;
    if (id < n) {
        int nodeTo = edgeNodeMappingTo[id];
        int nodeFrom = edgeNodeMappingFrom[id];
        if (hiprand_uniform(&states[blockIdx.x]) >= weights[id]) {
            atomicAdd(&outputs[nodeTo], outputs[nodeFrom]);
        }
        outputs[nodeTo] = 1/(1+exp(-1*(outputs[nodeTo]+bias[nodeTo])));
    }
}

__global__ void runGPUMultiplex(bool* activeEdges, float* weights, int* edgeNodeMappingTo, int* edgeNodeMappingFrom, float* outputs, float* bias, int offset, int n) {
    int id = blockIdx.x*blockDim.x+threadIdx.x + offset;
    if (id < n) {
        int nodeTo = edgeNodeMappingTo[id];
        int nodeFrom = edgeNodeMappingFrom[id];
        if (activeEdges[id]) {
            atomicAdd(&outputs[nodeTo], outputs[nodeFrom] * weights[id]);
        }
        outputs[nodeTo] = 1/(1+exp(-1*(outputs[nodeTo]+bias[nodeTo])));

        //outputs[nodeTo]
    }
}


/*
learningRate
momentum
target: Node-wise
weights: Edge-wise
outputs: Node-wise
edgeNodeMappingFrom: Edge-wise
edgeNodeMappingTo: Edge-wise
nodeRunCount: Node-wise
initialNodeRunCount: Node-wise
errors: Node-wise
deltas: Node-wise
bias: Node-wise
changes: Edge-wise
n
*/
__global__ void learnGPU(float learningRate,
                         float momentum,
                         float* weights,
                         float* outputs,
                         int* edgeNodeMappingFrom,
                         int* edgeNodeMappingTo,
                         float* errors,
                         float* deltas,
                         float* bias,
                         float* changes,
                         int offset,
                         int n,
                          float* buffer) {
    int id = blockIdx.x*blockDim.x+threadIdx.x+offset;
    if (id < offset+n && id>=0) {
        int nodeTo = edgeNodeMappingTo[id];
        int nodeFrom = edgeNodeMappingFrom[id];
        float output = outputs[nodeFrom];

        float& weight = weights[id];
        float delta = deltas[nodeTo];

        atomicAdd(&errors[nodeFrom], delta * weight);
        deltas[nodeFrom] = errors[nodeFrom] * output * (1-output);

        //atomicAdd(&nodeRunCount[nodeFrom], -1);
        //if (nodeRunCount[nodeFrom] == 0) {
        //bias[nodeTo] += learningRate * delta;
        //    nodeRunCount[nodeFrom] = initialNodeRunCount[nodeFrom];
        //}

        float& change = changes[id];
        change = (learningRate * delta * output)
                    + (momentum * change);
        weight += change;
    }
}

__global__ void learnGPUPost(float* deltas, float* bias, float learningRate, int offset, int n) {
    int id = blockIdx.x*blockDim.x+threadIdx.x+offset;
    if (id < offset+n && id>=0) {
        bias[id] = learningRate * deltas[id];
    }
}

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

float NN::trainGPU(vector<float> inputs, vector<float> target) {
    return trainGPU(inputs, target, 0.3, 0.1);
}

float NN::trainGPU(vector<float> inputs, vector<float> target, float learningRate, float momentum) {
    vector<float> results = runGPULauncher(inputs);

    int nodeSize = nodes.size();
    int outputSize = target.size();
    float errors[nodeSize];
    float deltas[nodeSize];

    memset(errors, 0, sizeof errors);
    memset(deltas, 0, sizeof deltas);

    float errorSum = 0;
    for (int i=1; i<=outputSize; ++i) {
        //cout << "output[1] = " << results[results.size()-i] << ", output[2] = " << std::round(results[results.size()-i]) << "\n";
        float output = results[results.size()-i];
        std::cout << "Output: " << output << "\n";
        errors[nodeSize-i] = target[outputSize-i] - output;
        deltas[nodeSize-i] = errors[nodeSize-i] * output * (1-output);
        errorSum += errors[nodeSize-i];
    }
    float error = abs(errorSum/outputSize);
    //std::cout << "Error: " << error << "\n";

    hipMemcpy(d_errors, &errors[0], nodeSize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_deltas, &deltas[0], nodeSize*sizeof(float), hipMemcpyHostToDevice);

    int offset = edges.size();
    int nodesOffset = nodes.size();

    float* d_buffer;
    //cudaMalloc(&d_buffer, offset*sizeof(float));
    for (int i=layers.size()-1; i>0; --i) {
        int N = layers[i].edges.size();
        int nodesN = layers[i].nodes.size();
        offset -= N;
        nodesOffset -= nodesN;
        //std::cout << "N["<<i<<"]: "<<N<<", offset["<<i<<"]: " << offset << "\n";
        int edgeCount = layers[i].edges.size();
        int nodeCount = layers[i].nodes.size();
        int blockSize = 512;// or 64?
        int gridSize = edgeCount/blockSize + 1;

        learnGPU<<<gridSize, blockSize>>>(learningRate,
                momentum,
                d_weights,
                d_outputs,
                d_edgeNodeMappingFrom,
                d_edgeNodeMappingTo,
                d_errors,
                d_deltas,
                d_bias,
                d_changes,
                offset,
                N,
                d_buffer);

        blockSize = 512;// or 64?
        gridSize = nodeCount/blockSize + 1;
        learnGPUPost<<<gridSize, blockSize>>>(d_deltas, d_bias, learningRate, nodesOffset, nodesN);
        //cudaCheckErrors("kernel");
    }
    /*float* buffer = (float*) (sizeof(float)*edges.size());
    cudaMemcpy(buffer, d_buffer, edges.size()*sizeof(float), cudaMemcpyDeviceToHost);
    for (int i=0; i<edges.size(); ++i) {
        std::cout << "buffer["<<i<<"] = " << buffer[i] << "\n";
    }*/
    return error;
}
vector<float> NN::runGPULauncher(vector<float>& inputs) {
    clock_t begin = clock();
    //cout << "Time ms: " << ((clock() - begin)/(double)CLOCKS_PER_SEC)*1000 << "\n";

    for (int i=inputs.size(); i<nodes.size(); ++i) {
        inputs.push_back(0);
    }

    hipMemcpy(d_outputs, &inputs[0], inputs.size()*sizeof(float), hipMemcpyHostToDevice);
    //cudaCheckErrors("copy");
    //cout << "Time Post Copy: " << ((clock() - begin)/(double)CLOCKS_PER_SEC)*1000 << "\n";

    int offset = 0;
    int nodeOffset = layers[0].nodes.size();
    //cout << "\nnodeCount: " << layers[0].nodes.size() << "\n";
    for (int i=1; i<layers.size(); ++i) {
        int edgeCount = layers[i].edges.size();
        int nodeCount = layers[i].nodes.size();
        //cout << "nodeCount: " << nodeCount << "\n";
        int gridSize, blockSize;
        blockSize = 512;// or 64?
        gridSize = edgeCount/blockSize + 1;
        runGPU<<<gridSize, blockSize>>>(d_weights, d_edgeNodeMappingTo, d_edgeNodeMappingFrom, d_outputs, d_bias, d_managementThreads, offset, offset+edgeCount);
        //runGPUProb<<<gridSize, blockSize>>>(d_weights, d_edgeNodeMappingTo, d_edgeNodeMappingFrom, d_outputs, d_bias, offset, offset+edgeCount, RNGStates);

        blockSize = 512;// or 64?
        gridSize = nodeCount/blockSize + 1;
        runGPUPost<<<gridSize, blockSize>>>(d_outputs, d_bias, nodeOffset, nodeOffset+nodeCount);

        //cout << "Time Post Kernel " << i << ": " << ((clock() - begin)/(double)CLOCKS_PER_SEC)*1000 << "\n";
        //cudaCheckErrors("kernel");
        offset += edgeCount;
        nodeOffset += nodeCount;
    }
    int outputLayerSize = layers[layers.size()-1].nodes.size();
    //cout << "\nOutput Size = " << outputLayerSize << "\n";
    float* outputs = (float*) malloc( sizeof(float)*outputLayerSize );
    //float* weights = (float*) malloc( sizeof(float)*edges.size() );

    hipMemcpy(outputs, d_outputs+(nodes.size()-outputLayerSize), outputLayerSize*sizeof(float), hipMemcpyDeviceToHost);
    //cudaMemcpy(weights, d_weights, edges.size()*sizeof(float), cudaMemcpyDeviceToHost);

    for (int i=0; i<edges.size(); ++i) {
    //    std::cout << "Weight["<<i<<"] = " << weights[i] << "\n";
    }

    //cout << "Time post copy output: " << ((clock() - begin)/(double)CLOCKS_PER_SEC)*1000 << "\n";
    vector<float> result(outputs, outputs + outputLayerSize);
    free(outputs);
    return result;
}

vector<float> NN::runCpu(vector<float>& inputs) {
    vector<float> result;
    while (inputs.size() < layers[0].nodes.size()) {
        inputs.push_back(0);
    }

    // Stage input vertices
    for (int i=0; i<inputs.size(); ++i) {
        layers[0].nodes[i]->output = inputs[i];
        result.push_back(inputs[i]);
    }

    int layerSize = layers.size();

    // Forward propegate each layer
    for (int i=1; i<layerSize; ++i) {
        vector<Vertex*> layerNodes = layers[i].nodes;
        for (int j=0; j<layerNodes.size(); ++j) {
            Vertex* node = layerNodes[j];
            float sum = node->bias;
            for (int inputIdx=0; inputIdx < node->inputs.size(); ++inputIdx) {
                Edge* inputEdge = node->inputs[inputIdx];
                sum += inputEdge->weight * inputEdge->from->output;
            }
            float outputValue = 1/(1+exp(-1*sum));
            node->output = outputValue;

            if (i==outputIdx) {
                result.push_back(outputValue);
            }
        }
    }
    return result;
}
void NN::print(ostream& output) {
//    output << "View at: http://www.webgraphviz.com/\n";
    output << "digraph G {\n";
    stringstream edges;
    for (int i=0; i<layers.size(); ++i) {
        output << "\tsubgraph cluster_" << i << " {\n"
             << "\t\tstyle=filled;\n"
             << "\t\tcolor=lightgrey;\n"
             << "\t\tnode [style=filled,color=white];\n";

        for (int j=0; j<layers[i].nodes.size(); ++j) {
            output << "\t\t\"" << layers[i].nodes[j]->index << "\"\n";
            for (int k=0; k<layers[i].nodes[j]->outputs.size(); k++) {
                edges << "\t\"" << layers[i].nodes[j]->index << "\" -> \"" << layers[i].nodes[j]->outputs[k]->to->index << "\";\n";
            }
        }

        output << "\t\tlabel = \"layer #" << i << "\";\n";
        output << "\t}\n";
    }
    output << edges.str();

    output << "}\n";
}

Vertex* NN::addVertex(std::vector<Vertex*> inputs, std::vector<Vertex*> outputs) {
    Vertex* vertex = new Vertex;
    vertex->index = nodeIdx++;
    vertex->bias = 0.0;
    vertex->error = 0.0;
    vertex->delta = 0.0;
    vertex->output = 0.0;

    vector<Edge*> edges;
    for (int i=0; i<inputs.size(); ++i) {
        edges.push_back(addEdge(inputs[i], vertex));
    }

    for (int i=0; i<outputs.size(); ++i) {
        edges.push_back(addEdge(vertex, outputs[i]));
    }

    nodes.push_back(vertex);
    int layerIdx = findLayer(vertex);

    if (layerIdx < (int)layers.size()) {
        layers[layerIdx].nodes.push_back(vertex);
    } else {
        Layer layer;
        layer.nodes.push_back(vertex);
        layers.insert(layers.begin()+layerIdx, layer);
    }
    vertex->layer = layerIdx;
    layers[layerIdx].edges.insert(layers[layerIdx].edges.end(), edges.begin(), edges.end());
    return vertex;
}

Vertex* NN::addVertex(int inputCount, int outputCount, int layer, bool useBasicLayering) {
    vector<Vertex*> inputs;
    vector<Vertex*> outputs;
    bool useBasicLayeringSave = useBasicLayering;
    useBasicLayering = true;
    for (int i=0; i<inputCount; ++i) {
        Layer* inputLayer = useBasicLayering ? &layers[layer-1] : &layers[random(0,layer-1)];
        int inputVertex = random(0, inputLayer->nodes.size()-1);
        inputs.push_back(inputLayer->nodes[inputVertex]);

        if (i==0) useBasicLayering=useBasicLayeringSave;
    }
    for (int i=0; i<outputCount; ++i) {
        Layer* outputLayer;
        if (useBasicLayering) {
            outputLayer = &layers[layer+1];
        }else{
            int outputLayerId = random(0,layer+1);
            if (outputLayerId == layer) {
                outputLayerId++;
            }
            outputLayer = &layers[outputLayerId];
        }
        int outputVertex = random(0, outputLayer->nodes.size()-1);
        outputs.push_back(outputLayer->nodes[outputVertex]);
    }
    return addVertex(inputs, outputs);
}

Vertex* NN::addVertex(int inputs, int outputs, bool isInput, bool isOutput, int layer, bool useStrictLayers) {
    Vertex* vertex = new Vertex;
    vertex->index = nodeIdx++;
    vertex->isOutput = isOutput;
    vertex->bias = 0.0;
    vertex->error = 0.0;
    vertex->delta = 0.0;
    vertex->output = 0.0;
    vector<Edge*> edges;

    inputs = inputs>0?inputs:inputs+1;
    int* edgeNodeMapping = (int*) malloc(sizeof(int)*inputs);

    if (isInput) {
        inputs--;
    }

    if (layer > 0) {
        Layer* inputLayer = &layers[layer-1];
        int inputVertex = random(0, inputLayer->nodes.size()-1);
        edges.push_back(addEdge(inputLayer->nodes[inputVertex], vertex));
        inputs--;
    }

    for (int i=0; i<inputs; ++i) {
        Layer* inputLayer = useStrictLayers ? &layers[layer-1] : &layers[random(0,layer-1)];
        int inputVertex = random(0, inputLayer->nodes.size()-1);
        edges.push_back(addEdge(inputLayer->nodes[inputVertex], vertex));
    }
    for (int i=0; i<outputs; ++i) {
        int outputVertex = random(inputSize, nodes.size());
    }

    nodes.push_back(vertex);
    int layerIdx = findLayer(vertex);

    if (isOutput && outputIdx != 0) {
        isOutput = false;
        layerIdx = outputIdx;
    }

    if (layerIdx < (int)layers.size() && !isOutput) {
        layers[layerIdx].nodes.push_back(vertex);
    } else {
        if (isOutput) {
            outputIdx = layers.size();
            layerIdx = outputIdx;
        }
        Layer layer;
        layer.nodes.push_back(vertex);
        layers.insert(layers.begin()+layerIdx, layer);
    }
    vertex->layer = layerIdx;
    layers[layerIdx].edges.insert(layers[layerIdx].edges.end(), edges.begin(), edges.end());
    return vertex;
}

// TODO: remove edge from layer
void NN::removeVertex() {
    int vertexIdx = random(0, nodes.size());
    Vertex* vertex = nodes[vertexIdx];
    for (int i=0; i<vertex->inputs.size(); ++i) {
        removeEdge(vertex->inputs[i]);
    }
    for (int i=0; i<vertex->outputs.size(); ++i) {
        removeEdge(vertex->outputs[i]);
    }
    nodes.erase(nodes.begin() + vertexIdx);
}

void NN::removeEdge(Edge* edge) {
    removeFromVector(edge->from->outputs, edge);
    removeFromVector(edge->to->inputs, edge);
    delete edge;
}

Edge* NN::addEdge(Vertex* from, Vertex* to) {
    return addEdge(from, to, false);
}

Edge* NN::addEdge(Vertex* from, Vertex* to, bool addLayer) {
    Edge *edge = new Edge;
    edge->to = to;
    edge->from = from;
    edge->weight = 0.1;
    edge->change = 0.0;
    to->inputs.push_back(edge);
    from->outputs.push_back(edge);
    edges.push_back(edge);
    if (addLayer) {
        layers[to->layer].edges.push_back(edge);
    }
    return edge;
}

template<typename T>
inline void removeFromVector(vector<T> & v, const T & item) {
    for(typename vector<T>::iterator iter = v.begin(); iter != v.end(); ++iter) {
        if(*iter == item) {
            v.erase(iter);
            break;
        }
    }
}

int NN::findLayer(Vertex* vertex) {
    int maxDepth = -1;
    for (int i=0; i<vertex->inputs.size(); ++i) {
        maxDepth = max(maxDepth, vertex->inputs[i]->from->layer);
    }
    vertex->layer = maxDepth+1;
    return maxDepth+1;
}

int random(int min, int max) {
    return rand()%(max-min + 1) + min;
}

float average(float average, float dataPoint) {
    static int N = 20;
    average -= average / N;
    average += dataPoint / N;
    return average;
}
